#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<vector>
#include<string>
#include<set>
#include <utility>
#include<unordered_set>
#include <functional>
#include<time.h>

#include"analysis.h"

unordered_set<string>generatedStringsUnique;
vector<pair<int, int>> reacheableVertices;
vector<pair<pair<int, int>, string>> possibleCFGStrings;

using namespace std;


bool is_member(const string& str)
{
    if (generatedStringsUnique.count(str)) {
        return true;
    }
    else {
        return false;
    }
}

void dfs(int from, int current, string path, const unordered_map<int, vector<pair<char, int>>>& edges) {
    // Add the current path to the result
    if (from != current) {
        possibleCFGStrings.push_back({ {from, current}, path });
    }

    // Explore neighbors
    if (edges.find(current) != edges.end()) {
        for (const auto& edge : edges.at(current)) {
            char label = edge.first;
            int to = edge.second;
            dfs(from, to, path + label, edges);  // Recursive DFS call
        }
    }
}

__device__ bool is_member_CUDA(const char* d_strings, int numStrings, int stringLength, const char* target) {

    int left = 0, right = numStrings - 1;

    while (left <= right) {
        int mid = left + (right - left) / 2;
        const char* midStr = &d_strings[mid * stringLength];

        int cmp = strncmp(midStr, target, stringLength);
        if (cmp == 0) return true;
        else if (cmp < 0) left = mid + 1;
        else right = mid - 1;
    }
    return false;
}

__global__ void checkMembershipKernel(
    const pair<int, int>* cfgVertices,
    const char* cfgStrings,
    int stringLength,
    bool* isMemberResults,
    int numEntries)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEntries) {
        const char* str = &cfgStrings[idx * stringLength];
        isMemberResults[idx] = is_member(str);
    }
}

void CFL_reachable(const CAnalysis& graphObj)
{
    unordered_map<int, vector<pair<char, int>>> edges = graphObj.getEdges();

    for (const auto& entry : edges)
    {
        int from = entry.first;
        dfs(from, from, "", edges);
    }

   /* for (const auto& entry : possibleCFGStrings) //Converting this in CUDA
    {
        if (is_member(entry.second)) {
            reacheableVertices.push_back({ entry.first.first, entry.first.second });
        }
    }*/

    int numEntries = possibleCFGStrings.size();
    int stringLength = 0;
    if (numEntries > 0) {
        stringLength = possibleCFGStrings[0].second.size();
    }
    // Allocate and copy cfgVertices and cfgStrings to GPU
    pair<int, int>* d_cfgVertices;
    char* d_cfgStrings;
    bool* d_isMemberResults;

    hipMalloc(&d_cfgVertices, numEntries * sizeof(pair<int, int>));
    hipMalloc(&d_cfgStrings, numEntries * stringLength * sizeof(char));
    hipMalloc(&d_isMemberResults, numEntries * sizeof(bool));

    pair<int, int>* h_cfgVertices = new pair<int, int>[numEntries];
    char* h_cfgStrings = new char[numEntries * stringLength];

    for (int i = 0; i < numEntries; ++i) {
        h_cfgVertices[i] = possibleCFGStrings[i].first;
        memcpy(&h_cfgStrings[i * stringLength], possibleCFGStrings[i].second.c_str(), stringLength);
    }

    hipMemcpy(d_cfgVertices, h_cfgVertices, numEntries * sizeof(pair<int, int>), hipMemcpyHostToDevice);
    hipMemcpy(d_cfgStrings, h_cfgStrings, numEntries * stringLength * sizeof(char), hipMemcpyHostToDevice);

    // Copy results back to CPU
    bool* h_isMemberResults = new bool[numEntries];
    hipMemcpy(h_isMemberResults, d_isMemberResults, numEntries * sizeof(bool), hipMemcpyDeviceToHost);

    // Process the results
    for (int i = 0; i < numEntries; ++i) {
        if (h_isMemberResults[i]) {
            reacheableVertices.push_back(h_cfgVertices[i]);
        }
    }

    // Free GPU memory
    hipFree(d_cfgVertices);
    hipFree(d_cfgStrings);
    hipFree(d_isMemberResults);

    delete[] h_cfgVertices;
    delete[] h_cfgStrings;
    delete[] h_isMemberResults;

}


int main() {

    cout << "Program Starts\n\n";
    string graphFile = "input_graph.txt";
    string CGFFile = "CFG.txt";

    //construct graph
    CAnalysis obj;
    obj.contructGraph(graphFile);

    //generating strings
    string start = "S";
    int maxDepth = 7;// Maximum recursion depth to limit the length of generated strings

    unordered_map<char, vector<string>> productions;  // Stores the production rules
    string startSymbol;  // Start symbol (usually 'S')

    obj.parseGrammar(CGFFile, productions, startSymbol);

    //vector<string> generated_All_Possible_Strings;
    //obj.generateStrings(productions, startSymbol, 0, maxDepth, generatedStrings);

    clock_t parse_start, parse_end;
    parse_start = clock();
    obj.generateStrings(productions, startSymbol, 0, maxDepth, generatedStringsUnique);
    parse_end = clock();

    printf("String Generation time : %4.6f sec\n",
        (double)((double)(parse_end - parse_start) / CLOCKS_PER_SEC));

    //run DFS

    /*cout << "Generated Strings:" << endl;
    for (const string& str : generatedStringsUnique) {
        cout << str << endl;
    }*/

    //DFS
    clock_t analysis_start, analysis_end;
    analysis_start = clock();
    CFL_reachable(obj);
    analysis_end = clock();

    printf("Analysis time : %4.6f sec\n",
        (double)((double)(analysis_end - analysis_start) / CLOCKS_PER_SEC));

    cout << "All vertices CFL-reachable from s: \n";
    for (const auto& pair : reacheableVertices) {
        cout << "(" << pair.first << ", " << pair.second << ")" << endl;
    }

    cout << "\nProgram Ends\n";


    return 0;
}
